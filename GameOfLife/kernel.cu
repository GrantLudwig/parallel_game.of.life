#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdint.h>
#include <cstdint>
#include <assert.h>
#include <algorithm>
#include <time.h>

typedef unsigned int uint;
typedef unsigned char uchar;
typedef unsigned short ushort;

__global__ void nextGenCuda(const uchar* lifeData, uint worldWidth, uint worldHeight, uchar* resultLifeData) {
	uint worldSize = worldWidth * worldHeight;

	for (uint cellId = __mul24(blockIdx.x, blockDim.x) + threadIdx.x; cellId < worldSize; cellId += blockDim.x * gridDim.x) {
		uint x = cellId % worldWidth;
		uint yAbs = cellId - x;
		uint xLeft = (x + worldWidth - 1) % worldWidth;
		uint xRight = (x + 1) % worldWidth;
		uint yAbsUp = (yAbs + worldSize - worldWidth) % worldSize;
		uint yAbsDown = (yAbs + worldWidth) % worldSize;

		uint aliveCells = lifeData[xLeft + yAbsUp] + lifeData[x + yAbsUp] + lifeData[xRight + yAbsUp]
			+ lifeData[xLeft + yAbs] + lifeData[xRight + yAbs]
			+ lifeData[xLeft + yAbsDown] + lifeData[x + yAbsDown] + lifeData[xRight + yAbsDown];

		resultLifeData[x + yAbs] = aliveCells == 3 || (aliveCells == 2 && lifeData[x + yAbs]) ? 1 : 0;
	}
}

void nextGenCpu(const uchar* lifeData, uint worldWidth, uint worldHeight, uchar* resultLifeData) {
	uint worldSize = worldWidth * worldHeight;

	for (uint cellId = 0; cellId < worldSize; cellId++) {
		uint x = cellId % worldWidth;
		uint yAbs = cellId - x;
		uint xLeft = (x + worldWidth - 1) % worldWidth;
		uint xRight = (x + 1) % worldWidth;
		uint yAbsUp = (yAbs + worldSize - worldWidth) % worldSize;
		uint yAbsDown = (yAbs + worldWidth) % worldSize;

		uint aliveCells = lifeData[xLeft + yAbsUp] + lifeData[x + yAbsUp] + lifeData[xRight + yAbsUp]
			+ lifeData[xLeft + yAbs] + lifeData[xRight + yAbs]
			+ lifeData[xLeft + yAbsDown] + lifeData[x + yAbsDown] + lifeData[xRight + yAbsDown];

		resultLifeData[x + yAbs] = aliveCells == 3 || (aliveCells == 2 && lifeData[x + yAbs]) ? 1 : 0;
	}
}

void runGameCpu(uchar*& lifeData, uchar*& lifeDataBuffer, size_t worldWidth, size_t worldHeight, size_t iterationsCount) {
	for (size_t i = 0; i < iterationsCount; i++) {
		nextGenCpu(lifeData, worldWidth, worldHeight, lifeDataBuffer);
		std::swap(lifeData, lifeDataBuffer);
	}
}

void runGameCuda(uchar*& d_lifeData, uchar*& d_lifeDataBuffer, size_t worldWidth, size_t worldHeight, size_t iterationsCount, ushort threadsCount) {
	assert((worldWidth * worldHeight) % threadsCount == 0);
	size_t reqBlocksCount = (worldWidth * worldHeight) / threadsCount;
	ushort blocksCount = (ushort)std::min((size_t)32768, reqBlocksCount);

	for (size_t i = 0; i < iterationsCount; i++) {
		nextGenCuda << <blocksCount, threadsCount >> > (d_lifeData, worldWidth, worldHeight, d_lifeDataBuffer);
		std::swap(d_lifeData, d_lifeDataBuffer);
	}
}

int main()
{
	// setup data
	// ==========
	const size_t WORLD_WIDTH = 10000; // 10,000
	const size_t WORLD_HEIGHT = 10000; // 10,000
	const size_t NUM_ITERATIONS = 10000000; // 10,000,000
	const ushort NUM_THREADS = 5000; //GPU threads

	size_t size_data = sizeof(uchar) * WORLD_WIDTH * WORLD_HEIGHT;

	// serial using CPU
	// ===================

	printf("Serial: Setting up data for game\n");

	uchar *lifeData = reinterpret_cast<uchar *>(malloc(size_data));
	uchar *lifeDataBuffer = reinterpret_cast<uchar *>(malloc(size_data));

	memset(lifeData, 0, size_data); // initilize everything to 0

	lifeData[3 * WORLD_WIDTH + 3] = 1;
	lifeData[4 * WORLD_WIDTH + 3] = 1;
	lifeData[5 * WORLD_WIDTH + 3] = 1;

	printf("Serial: Starting game\n");
	clock_t begin = clock();

	runGameCpu(lifeData, lifeData, WORLD_WIDTH, WORLD_HEIGHT, NUM_ITERATIONS);

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Serial: Game complete\n");
	printf("Took %f sec for %zu iterations\n\n", time_spent, NUM_ITERATIONS);

	free(lifeData);
	free(lifeDataBuffer);

	// parallel using CUDA
	// ===================

	printf("Parallel: Setting up data for game\n");

	// init host arrays
	uchar *h_lifeData = reinterpret_cast<uchar *>(malloc(size_data));
	uchar *h_lifeDataBuffer = reinterpret_cast<uchar *>(malloc(size_data));

	for (size_t i = 0; i < WORLD_WIDTH; i++) {

		for (size_t j = 0; j < WORLD_HEIGHT; j++) {
			h_lifeData[j * WORLD_WIDTH + i] = 0;
		}
	}

	h_lifeData[3 * WORLD_WIDTH + 3] = 1;
	h_lifeData[4 * WORLD_WIDTH + 3] = 1;
	h_lifeData[5 * WORLD_WIDTH + 3] = 1;

	//init device arrays
	uchar *d_lifeData, *d_lifeDataBuffer;
	hipMalloc(reinterpret_cast<void **>(&d_lifeData), size_data);
	hipMalloc(reinterpret_cast<void **>(&d_lifeDataBuffer), size_data);

	hipMemcpy(d_lifeData, h_lifeData, size_data, hipMemcpyHostToDevice);
	hipMemcpy(d_lifeDataBuffer, h_lifeDataBuffer, size_data, hipMemcpyHostToDevice);

	printf("Parallel: Starting game\n");
	begin = clock();
	
	runGameCuda(d_lifeData, d_lifeDataBuffer, WORLD_HEIGHT, WORLD_HEIGHT, NUM_ITERATIONS, NUM_THREADS);

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Parallel: Game complete\n");
	printf("Took %f sec for %zu iterations\n", time_spent, NUM_ITERATIONS);

	hipMemcpy(h_lifeDataBuffer, d_lifeData, size_data, hipMemcpyDeviceToHost); // to get final

	free(h_lifeData);
	free(h_lifeDataBuffer);

	hipFree(d_lifeData);
	hipFree(d_lifeDataBuffer);

	return 0;
}

/*for (size_t i = 0; i < WORLD_WIDTH; i++) {

		for (size_t j = 0; j < WORLD_WIDTH; j++) {
			if (h_lifeData[j * WORLD_WIDTH + i] == 1)
				printf("*");
			else
				printf("_");
		}
		printf("\n");
	}

	printf("\n");*/